#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <limits>
#include <sstream>
#include <string>

__global__ void kernel(float *Wa, float *Wb, float *Wc, float *Ma, float *Mb,
                       float *Mc, int maxNumOfIters, float nueAB, float nueC,
                       float tol, int n, int p, int seed, float *ret,
                       float *success) {

  const int threadId = threadIdx.x;
  const int blockId = blockIdx.x;

  const int nn = n * n;
  const int matrixSizeW = nn * p * sizeof(float);
  const int matrixSizeABC = nn * sizeof(float);
  float *myWa, *myWb, *myWc;
  myWa = (float *)malloc(matrixSizeW);
  myWb = (float *)malloc(matrixSizeW);
  myWc = (float *)malloc(matrixSizeW);
  memcpy(myWa, Wa, matrixSizeW);
  memcpy(myWb, Wb, matrixSizeW);
  memcpy(myWc, Wc, matrixSizeW);
  float *a, *b, *c, *aStar, *bStar, *cStar, *cDiff;
  a = (float *)malloc(matrixSizeABC);
  b = (float *)malloc(matrixSizeABC);
  c = (float *)malloc(matrixSizeABC);
  aStar = (float *)malloc(p);
  bStar = (float *)malloc(p);
  cStar = (float *)malloc(p);
  cDiff = (float *)malloc(matrixSizeABC);

  int startVal = abs((seed + blockId * 3 + threadId * 7 +
                      ((int)clock() / 10000000) % INT_MAX + (int)(*ret) * 23) %
                     INT_MAX);

  // printf("seed %i \n", startVal);

  hiprandState_t state;
  hiprand_init(startVal, threadId + blockId, 11, &state);

  int inTolCount = 0;

  for (int iter = 0; iter < maxNumOfIters; iter++) {

    float normA = 0.0;
    float normB = 0.0;

    // a und b zufällig initialisieren
    for (int i = 0; i < nn; i++) {
      a[i] = 1.0 - (float)hiprand(&state) / (float)INT_MAX;
      b[i] = 1.0 - (float)hiprand(&state) / (float)INT_MAX;
      normA += a[i] * a[i];
      normB += b[i] * b[i];
    }

    normA = 1.0 / sqrt(normA);
    normB = 1.0 / sqrt(normB);

    // normieren a und b:
    for (int i = 0; i < nn; i++) {
      a[i] *= normA;
      b[i] *= normB;
    }

    // korrektes c
    for (int i = 0; i < n; i++) {
      for (int j = 0; j < n; j++) {
        c[i * n + j] = 0.0;
        for (int k = 0; k < n; k++) {
          c[i * n + j] += a[i * n + k] * b[k * n + j];
        }
      }
    }

    // c* = a* x b*
    for (int i = 0; i < p; i++) {
      aStar[i] = 0.0;
      bStar[i] = 0.0;
      for (int j = 0; j < nn; j++) {
        aStar[i] += myWa[nn * i + j] * a[j];
        bStar[i] += myWb[nn * i + j] * b[j];
      }
      cStar[i] = aStar[i] * bStar[i];
    }

    float err = 0.0;

    // c_wave - c  .. Fehler in c
    for (int i = 0; i < nn; i++) {
      float cWave = 0.0;
      for (int k = 0; k < p; k++) {
        cWave += myWc[p * i + k] * cStar[k];
      }
      cDiff[i] = cWave - c[i];
      err += cDiff[i] * cDiff[i];
    }

    err = sqrt(err);
    if (iter % 1000000 == 0 && iter > 0) {
      printf("block %i, thread %i, iter %i err = %f\n", blockId, threadId, iter,
             err);
    }

    if (isnan(err) || isinf(err) || err > 1000 || *success > 0) {
      return;
    }

    if (err < tol) {
      inTolCount++;
      if (inTolCount > 100) {
        *success = 1.0;
        printf("beendet durch block %i, thread %i mit err = %f \n", blockId,
               threadId, err);
        return;
      }
    } else {
      inTolCount = 0;
    }

    // Korrektur Wa und Wb
    for (int i = 0; i < p; i++) {
      float WcTCDiff = 0.0;
      for (int j = 0; j < nn; j++) {
        WcTCDiff += myWc[i + j * p] * cDiff[j];
      }
      float WCBStar = WcTCDiff * bStar[i] * nueAB;
      float WCAStar = WcTCDiff * aStar[i] * nueAB;
      for (int j = 0; j < nn; j++) {
        myWa[i * nn + j] -= WCBStar * a[j] * Ma[i * nn + j];
        myWb[i * nn + j] -= WCAStar * b[j] * Mb[i * nn + j];
      }
    }

    // Korrektur Wc
    for (int i = 0; i < nn; i++) {
      float CDiffNue = cDiff[i] * nueC;
      for (int j = 0; j < p; j++) {
        myWc[i * p + j] -= CDiffNue * cStar[j] * Ma[i * p + j];
      }
    }
  } // iter
} // kernel

float runBackpropOnGPU(float *Wa, float *Wb, float *Wc, float *Ma, float *Mb,
                       float *Mc, int maxNumIters, float nueAB, float nueC,
                       float tol, int n, int p, int seed) {
  std::cout << "runBackpropOnGPU n = " << n << ", p = " << p << '\n';

  int nn = n * n;

  float *WaGPU, *WbGPU, *WcGPU;
  float *MaGPU, *MbGPU, *McGPU;
  float err, *errDevice, *successDevice;
  float success = -1.0;

  hipMalloc(&errDevice, sizeof(float));
  hipMalloc(&successDevice, sizeof(float));

  hipMalloc(&WaGPU, nn * p * sizeof(float));
  hipMalloc(&WbGPU, nn * p * sizeof(float));
  hipMalloc(&WcGPU, nn * p * sizeof(float));
  hipMalloc(&MaGPU, nn * p * sizeof(float));
  hipMalloc(&MbGPU, nn * p * sizeof(float));
  hipMalloc(&McGPU, nn * p * sizeof(float));

  hipMemcpy(successDevice, &success, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(WaGPU, Wa, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(WbGPU, Wb, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(WcGPU, Wc, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(MaGPU, Ma, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(MbGPU, Mb, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(McGPU, Mc, nn * p * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockGrid(36);
  dim3 threadGrid(32);
  kernel<<<blockGrid, threadGrid>>>(WaGPU, WbGPU, WcGPU, MaGPU, MbGPU, McGPU,
                                    maxNumIters, nueAB, nueC, tol, n, p, seed,
                                    errDevice, successDevice);

  hipMemcpy(&err, errDevice, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&success, successDevice, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wa, WaGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wb, WbGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wc, WcGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Ma, MaGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Mb, MbGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Mc, McGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(errDevice);
  hipFree(successDevice);
  hipFree(WaGPU);
  hipFree(WbGPU);
  hipFree(WcGPU);
  hipFree(MaGPU);
  hipFree(MbGPU);
  hipFree(McGPU);
  hipDeviceReset();

  return success;
}
