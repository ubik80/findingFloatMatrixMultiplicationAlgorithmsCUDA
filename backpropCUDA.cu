#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <iostream>

// memory allocation, remember pointers for cleanup later
__device__ float *mallocGb(int numOfFloats, float **garbageDump,
                           int garbageCounter) {
  garbageDump[garbageCounter] = (float *)malloc(numOfFloats * sizeof(float));
  garbageCounter++;
  return garbageDump[garbageCounter - 1];
}

// free all allocated
__device__ void freeGb(float **garbageDump, int garbageCounter) {
  for (int i = 0; i < garbageCounter; i++) {
    free(garbageDump[i]);
  }
}

__device__ void lock(int *mutex) {
  while (atomicCAS(mutex, 0, 1) != 0) {
  };
}

__device__ void unlock(int *mutex) { atomicExch(mutex, 0); }

__global__ void kernel(float *Wa, float *Wb, float *Wc, int maxNumOfIters,
                       float nueAB, float nueC, float tol, int n, int p,
                       int seed, int *killSignal, int *mutex, float *minError) {
  float *garbageDump[10];
  int garbageCounter = 0;

  const int threadId = threadIdx.x;
  const int blockId = blockIdx.x;
  const int nn = n * n;

  float *myWa = (float *)mallocGb(nn * p, garbageDump, garbageCounter);
  float *myWb = (float *)mallocGb(nn * p, garbageDump, garbageCounter);
  float *myWc = (float *)mallocGb(nn * p, garbageDump, garbageCounter);
  memcpy(myWa, Wa, nn * p * sizeof(float));
  memcpy(myWb, Wb, nn * p * sizeof(float));
  memcpy(myWc, Wc, nn * p * sizeof(float));
  float *a = (float *)mallocGb(nn, garbageDump, garbageCounter);
  float *b = (float *)mallocGb(nn, garbageDump, garbageCounter);
  float *c = (float *)mallocGb(nn, garbageDump, garbageCounter);
  float *aStar = (float *)mallocGb(p, garbageDump, garbageCounter);
  float *bStar = (float *)mallocGb(p, garbageDump, garbageCounter);
  float *cStar = (float *)mallocGb(p, garbageDump, garbageCounter);
  float *cDiff = (float *)mallocGb(nn, garbageDump, garbageCounter);

  int startVal = abs((seed + blockId * 3 + threadId * 7 +
                      ((int)clock() / 10000000) % INT_MAX) %
                     INT_MAX);
  hiprandState_t state;
  hiprand_init(startVal, threadId + blockId, 11, &state);

  int inTolCount = 0; // counts interations with err < tol
  for (int iter = 0; iter < maxNumOfIters; iter++) {

    float normA = 0.0;
    float normB = 0.0;

    // randomly set a and b
    do {
      for (int i = 0; i < nn; i++) {
        a[i] = 1.0 - (float)hiprand(&state) / (float)INT_MAX;
        b[i] = 1.0 - (float)hiprand(&state) / (float)INT_MAX;
        normA += a[i] * a[i];
        normB += b[i] * b[i];
      }
    } while (normA < 0.1 or normB < 0.1);

    normA = 1.0 / sqrt(normA);
    normB = 1.0 / sqrt(normB);

    // scale a and b to length 1
    for (int i = 0; i < nn; i++) {
      a[i] *= normA;
      b[i] *= normB;
    }

    // calculate c (mat(c)=mat(a)*mat(b))
    for (int i = 0; i < n; i++) {
      for (int j = 0; j < n; j++) {
        c[i * n + j] = 0.0;
        for (int k = 0; k < n; k++) {
          c[i * n + j] += a[i * n + k] * b[k * n + j];
        }
      }
    }

    // c* = a* x b*
    for (int i = 0; i < p; i++) {
      aStar[i] = 0.0;
      bStar[i] = 0.0;
      for (int j = 0; j < nn; j++) {
        aStar[i] += myWa[nn * i + j] * a[j];
        bStar[i] += myWb[nn * i + j] * b[j];
      }
      cStar[i] = aStar[i] * bStar[i];
    }

    float err = 0.0;

    // c_wave - c  .. Fehler in c
    for (int i = 0; i < nn; i++) {
      float cWave = 0.0;
      for (int k = 0; k < p; k++) {
        cWave += myWc[p * i + k] * cStar[k];
      }
      cDiff[i] = cWave - c[i];
      err += cDiff[i] * cDiff[i];
    }

    err = sqrt(err);

    if (isnan(err) || isinf(err) || isinf(-err) || err > 10000 ||
        *killSignal > 0) {
      freeGb(garbageDump, garbageCounter);
      return;
    }

    if (iter % max((int)(maxNumOfIters / 5), 1000) == 0 && iter > 0) {
      printf("kernel: block %i, thread %i, iter %i err = %f\n", blockId,
             threadId, iter, err);
    }

    // innovate Wa and Wb
    for (int i = 0; i < p; i++) {
      float WcTCDiff = 0.0;
      for (int j = 0; j < nn; j++) {
        WcTCDiff += myWc[i + j * p] * cDiff[j];
      }
      float WCBStar = WcTCDiff * bStar[i] * nueAB;
      float WCAStar = WcTCDiff * aStar[i] * nueAB;
      for (int j = 0; j < nn; j++) {
        myWa[i * nn + j] -= WCBStar * a[j];
        myWb[i * nn + j] -= WCAStar * b[j];
      }
    }

    // innovate Wc
    for (int i = 0; i < nn; i++) {
      float CDiffNue = cDiff[i] * nueC;
      for (int j = 0; j < p; j++) {
        myWc[i * p + j] -= CDiffNue * cStar[j];
      }
    }

    if (err < tol) {
      inTolCount++;
      if (inTolCount > 10000) {
        lock(mutex);
        if (*killSignal > 0) {
          unlock(mutex);
          freeGb(garbageDump, garbageCounter);
          return;
        }
        atomicAdd(killSignal, 1);
        printf("kernel: Solved by block %i, thread %i with err = %f.\n ",
               blockId, threadId, err);
        for (int i = 0; i < nn * p; i++) {
          Wa[i] = myWa[i];
          Wb[i] = myWb[i];
          Wc[i] = myWc[i];
        }
        *minError = err;
        unlock(mutex);
        freeGb(garbageDump, garbageCounter);
        return;
      }
    } else {
      inTolCount = 0;
    }
  } // iter
} // kernel()

// memory operations and starting of kernels on GPU
float runBackpropOnGPU(float *Wa, float *Wb, float *Wc, int maxNumIters,
                       float nueAB, float nueC, float tol, int n, int p,
                       int seed, int blocks, int threads) {

  std::cout << "runBackpropOnGPU: n = " << n << ", p = " << p << '\n';
  std::cout << "runBackpropOnGPU: blocks = " << blocks
            << ", threads = " << threads << '\n';

  int nn = n * n;

  size_t grantedMemSize;
  size_t demandedMemSize =
      (nn * p * 3 + nn * 4 + p * 3) * sizeof(float) * blocks * threads * 2;
  hipDeviceGetLimit(&grantedMemSize, hipLimitMallocHeapSize);
  hipDeviceSetLimit(hipLimitMallocHeapSize,
                     max(grantedMemSize, demandedMemSize));
  hipDeviceGetLimit(&grantedMemSize, hipLimitMallocHeapSize);
  std::cout << "runBackpropOnGPU: demandedMemSize = " << demandedMemSize
            << '\n';
  std::cout << "runBackpropOnGPU: grantedMemSize =  " << grantedMemSize << '\n';

  if (grantedMemSize < demandedMemSize)
    return FLT_MAX; // mem. allocation declined

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel), hipFuncCachePreferL1);

  float minError = FLT_MAX;
  float *WaGPU, *WbGPU, *WcGPU, *minErrorGPU;
  int *killSignal, *mutex;

  hipMalloc(&killSignal, sizeof(int));
  hipMemset(killSignal, 0, sizeof(int));
  hipMalloc(&mutex, sizeof(int));
  hipMemset(mutex, 0, sizeof(int));
  hipMalloc(&minErrorGPU, sizeof(float));
  hipMemcpy(minErrorGPU, &minError, sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&WaGPU, nn * p * sizeof(float));
  hipMemcpy(WaGPU, Wa, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&WbGPU, nn * p * sizeof(float));
  hipMemcpy(WbGPU, Wb, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&WcGPU, nn * p * sizeof(float));
  hipMemcpy(WcGPU, Wc, nn * p * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockGrid(blocks);
  dim3 threadGrid(threads);
  kernel<<<blockGrid, threadGrid>>>(WaGPU, WbGPU, WcGPU, maxNumIters, nueAB,
                                    nueC, tol, n, p, seed, killSignal, mutex,
                                    minErrorGPU);

  hipMemcpy(&minError, minErrorGPU, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wa, WaGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wb, WbGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wc, WcGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(killSignal);
  hipFree(mutex);
  hipFree(minErrorGPU);
  hipFree(WaGPU);
  hipFree(WbGPU);
  hipFree(WcGPU);

  hipDeviceReset();

  std::cout << "runBackpropOnGPU: finished" << '\n';
  return minError;
}
