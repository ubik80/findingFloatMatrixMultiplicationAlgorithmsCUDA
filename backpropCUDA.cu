#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <limits>
#include <sstream>
#include <string>

void checkForCudaError(int line) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
    fprintf(stderr, "cudaCheckError() failed in line %i:\t%s\n", line,
            hipGetErrorString(err));
}

__device__ void lock(int *mutex){
  while (atomicCAS(mutex,0,1)!=0);
}

__device__ void unlock(int* mutex){
  atomicExch(mutex,0);
}

__global__ void kernel(float *Wa, float *Wb, float *Wc, float *Ma, float *Mb,
                       float *Mc, int maxNumOfIters, float nueAB, float nueC,
                       float tol, int n, int p, int seed, float *finalError,
                       int *mutex, int* killSignal) {

  const int threadId = threadIdx.x;
  const int blockId = blockIdx.x;

  const int nn = n * n;
  const int matrixSizeW = nn * p * sizeof(float);
  const int matrixSizeABC = nn * sizeof(float);
  float *myWa, *myWb, *myWc;
  myWa = (float *)malloc(matrixSizeW);
  myWb = (float *)malloc(matrixSizeW);
  myWc = (float *)malloc(matrixSizeW);
  memcpy(myWa, Wa, matrixSizeW);
  memcpy(myWb, Wb, matrixSizeW);
  memcpy(myWc, Wc, matrixSizeW);
  float *a, *b, *c, *aStar, *bStar, *cStar, *cDiff;
  a = (float *)malloc(matrixSizeABC);
  b = (float *)malloc(matrixSizeABC);
  c = (float *)malloc(matrixSizeABC);
  aStar = (float *)malloc(p);
  bStar = (float *)malloc(p);
  cStar = (float *)malloc(p);
  cDiff = (float *)malloc(matrixSizeABC);

  int startVal = abs((seed + blockId * 3 + threadId * 7 +
                      ((int)clock() / 10000000) % INT_MAX) %
                     INT_MAX);

  hiprandState_t state;
  hiprand_init(startVal, threadId + blockId, 11, &state);

  int inTolCount = 0;

  for (int iter = 0; iter < maxNumOfIters; iter++) {

    float normA = 0.0;
    float normB = 0.0;

    // a und b zufällig initialisieren
    for (int i = 0; i < nn; i++) {
      a[i] = 1.0 - (float)hiprand(&state) / (float)INT_MAX;
      b[i] = 1.0 - (float)hiprand(&state) / (float)INT_MAX;
      normA += a[i] * a[i];
      normB += b[i] * b[i];
    }

    normA = 1.0 / sqrt(normA);
    normB = 1.0 / sqrt(normB);

    // normieren a und b:
    for (int i = 0; i < nn; i++) {
      a[i] *= normA;
      b[i] *= normB;
    }

    // korrektes c
    for (int i = 0; i < n; i++) {
      for (int j = 0; j < n; j++) {
        c[i * n + j] = 0.0;
        for (int k = 0; k < n; k++) {
          c[i * n + j] += a[i * n + k] * b[k * n + j];
        }
      }
    }

    // c* = a* x b*
    for (int i = 0; i < p; i++) {
      aStar[i] = 0.0;
      bStar[i] = 0.0;
      for (int j = 0; j < nn; j++) {
        aStar[i] += myWa[nn * i + j] * a[j];
        bStar[i] += myWb[nn * i + j] * b[j];
      }
      cStar[i] = aStar[i] * bStar[i];
    }

    float err = 0.0;

    // c_wave - c  .. Fehler in c
    for (int i = 0; i < nn; i++) {
      float cWave = 0.0;
      for (int k = 0; k < p; k++) {
        cWave += myWc[p * i + k] * cStar[k];
      }
      cDiff[i] = cWave - c[i];
      err += cDiff[i] * cDiff[i];
    }

    err = sqrt(err);
    if (iter % (maxNumOfIters / 10) == 0 && iter > 0) {
      printf("block %i, thread %i, iter %i err = %f\n", blockId, threadId, iter,
             err);
    }

    if (isnan(err) || isinf(err) || err > 1000 || *killSignal == 1) {
      return;
    }

    if (err < tol) {
      inTolCount++;
      if (inTolCount > 100) {
        lock(mutex);
        if(*killSignal == 1){
          unlock(mutex);
          return;
        }
        if(err < *finalError){
          *finalError = err;
          *killSignal = 1;

          for(int i = 0; i < nn*p; i++){
            Wa[i] = myWa[i];
            Wb[i] = myWb[i];
            Wc[i] = myWc[i];
          }

          printf("beendet durch block %i, thread %i mit err = %f \n", blockId,
                  threadId, err);
          unlock(mutex);
          return;
        }
        unlock(mutex);
        return;
      }
    }
    else {
      inTolCount = 0;
    }

    // Korrektur Wa und Wb
    for (int i = 0; i < p; i++) {
      float WcTCDiff = 0.0;
      for (int j = 0; j < nn; j++) {
        WcTCDiff += myWc[i + j * p] * cDiff[j];
      }
      float WCBStar = WcTCDiff * bStar[i] * nueAB;
      float WCAStar = WcTCDiff * aStar[i] * nueAB;
      for (int j = 0; j < nn; j++) {
        myWa[i * nn + j] -= WCBStar * a[j] * Ma[i * nn + j];
        myWb[i * nn + j] -= WCAStar * b[j] * Mb[i * nn + j];
      }
    }

    // Korrektur Wc
    for (int i = 0; i < nn; i++) {
      float CDiffNue = cDiff[i] * nueC;
      for (int j = 0; j < p; j++) {
        myWc[i * p + j] -= CDiffNue * cStar[j] * Ma[i * p + j];
      }
    }
  } // iter
} // kernel

float runBackpropOnGPU(float *Wa, float *Wb, float *Wc, float *Ma, float *Mb,
                       float *Mc, int maxNumIters, float nueAB, float nueC,
                       float tol, int n, int p, int seed, int blocks, int threads) {
  std::cout << "runBackpropOnGPU n = " << n << ", p = " << p << '\n';

  int nn = n * n;

  float *WaGPU, *WbGPU, *WcGPU;
  float *MaGPU, *MbGPU, *McGPU;
  float *finalErrorDevice;
  float finalError = tol + 1.0;
  int *mutex, *killSignal;

  hipMalloc(&mutex, sizeof(int));
  hipMemset(mutex, 0, sizeof(int));
  hipMalloc(&killSignal, sizeof(int));
  hipMemset(killSignal, 0, sizeof(int));

  hipMalloc(&finalErrorDevice, sizeof(float));
  hipMalloc(&WaGPU, nn * p * sizeof(float));
  hipMalloc(&WbGPU, nn * p * sizeof(float));
  hipMalloc(&WcGPU, nn * p * sizeof(float));
  hipMalloc(&MaGPU, nn * p * sizeof(float));
  hipMalloc(&MbGPU, nn * p * sizeof(float));
  hipMalloc(&McGPU, nn * p * sizeof(float));

  hipMemcpy(finalErrorDevice, &finalError, sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(WaGPU, Wa, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(WbGPU, Wb, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(WcGPU, Wc, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(MaGPU, Ma, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(MbGPU, Mb, nn * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(McGPU, Mc, nn * p * sizeof(float), hipMemcpyHostToDevice);

  checkForCudaError(186);

  dim3 blockGrid(blocks);
  dim3 threadGrid(threads);
  kernel<<<blockGrid, threadGrid>>>(WaGPU, WbGPU, WcGPU, MaGPU, MbGPU, McGPU,
                                    maxNumIters, nueAB, nueC, tol, n, p, seed,
                                    finalErrorDevice, mutex, killSignal);

  checkForCudaError(194);

  hipMemcpy(&finalError, finalErrorDevice, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wa, WaGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wb, WbGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Wc, WcGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Ma, MaGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Mb, MbGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(Mc, McGPU, nn * p * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(killSignal);
  hipFree(mutex);
  hipFree(finalErrorDevice);
  hipFree(WaGPU);
  hipFree(WbGPU);
  hipFree(WcGPU);
  hipFree(MaGPU);
  hipFree(MbGPU);
  hipFree(McGPU);
  hipDeviceReset();

  checkForCudaError(215);

  return finalError;
}
